#include "hip/hip_runtime.h"
#include "mandelbrot/hw2.hh" 
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
__global__ void render(char *out, const int width, const int height, const int max_iter) 
{
  int x_dim = blockIdx.x*blockDim.x + threadIdx.x;
  int y_dim = blockIdx.y*blockDim.y + threadIdx.y;
  int index = 3*width*y_dim + x_dim*3;
  float x_origin = ((float) x_dim/width)*3.25 - 2;
  float y_origin = ((float) y_dim/width)*2.5 - 1.25;

  float x = 0.0;
  float y = 0.0;

  int iteration = 0;
  while(x*x + y*y <= 4 && iteration < max_iter) 
  {
    float xtemp = x*x - y*y + x_origin;
    y = 2*x*y + y_origin;
    x = xtemp;
    iteration++;
  }

  if(iteration == max_iter) 
  {
    out[index] = 0;
    out[index + 1] = 0;
    out[index + 2] = 0;
  } 
  else 
  {
    out[index] = iteration;
    out[index + 1] = iteration;
    out[index + 2] = iteration;
  }
}


/****************************************************************************
    bmp.c - read and write bmp images.
    Distributed with Xplanet.
    Copyright (C) 2002 Hari Nair <hari@alumni.caltech.edu>

    This program is free software; you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation; either version 2 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program; if not, write to the Free Software
    Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
****************************************************************************/

struct BMPHeader
{
    char bfType[2];       /* "BM" */
    int bfSize;           /* Size of file in bytes */
    int bfReserved;       /* set to 0 */
    int bfOffBits;        /* Byte offset to actual bitmap data (= 54) */
    int biSize;           /* Size of BITMAPINFOHEADER, in bytes (= 40) */
    int biWidth;          /* Width of image, in pixels */
    int biHeight;         /* Height of images, in pixels */
    short biPlanes;       /* Number of planes in target device (set to 1) */
    short biBitCount;     /* Bits per pixel (24 in this case) */
    int biCompression;    /* Type of compression (0 if no compression) */
    int biSizeImage;      /* Image size, in bytes (0 if no compression) */
    int biXPelsPerMeter;  /* Resolution in pixels/meter of display device */
    int biYPelsPerMeter;  /* Resolution in pixels/meter of display device */
    int biClrUsed;        /* Number of colors in the color table (if 0, use
                             maximum allowed by biBitCount) */
    int biClrImportant;   /* Number of important colors.  If 0, all colors
                             are important */
};

int read_bmp(const char *filename, int *width, int *height, unsigned char *rgb)
{
    fprintf(stderr, "Sorry, reading of .bmp files isn't supported yet.\n");
    return(0);
}

int write_bmp(const char *filename, int width, int height, char *rgb)
{
    int i, j, ipos;
    int bytesPerLine;
    unsigned char *line;

    FILE *file;
    struct BMPHeader bmph;

    /* The length of each line must be a multiple of 4 bytes */

    bytesPerLine = (3 * (width + 1) / 4) * 4;

    strncpy(bmph.bfType, "BM", 2);
    bmph.bfOffBits = 54;
    bmph.bfSize = bmph.bfOffBits + bytesPerLine * height;
    bmph.bfReserved = 0;
    bmph.biSize = 40;
    bmph.biWidth = width;
    bmph.biHeight = height;
    bmph.biPlanes = 1;
    bmph.biBitCount = 24;
    bmph.biCompression = 0;
    bmph.biSizeImage = bytesPerLine * height;
    bmph.biXPelsPerMeter = 0;
    bmph.biYPelsPerMeter = 0;
    bmph.biClrUsed = 0;
    bmph.biClrImportant = 0;

    file = fopen (filename, "wb");
    if (file == NULL) return(0);

    fwrite(&bmph.bfType, 2, 1, file);
    fwrite(&bmph.bfSize, 4, 1, file);
    fwrite(&bmph.bfReserved, 4, 1, file);
    fwrite(&bmph.bfOffBits, 4, 1, file);
    fwrite(&bmph.biSize, 4, 1, file);
    fwrite(&bmph.biWidth, 4, 1, file);
    fwrite(&bmph.biHeight, 4, 1, file);
    fwrite(&bmph.biPlanes, 2, 1, file);
    fwrite(&bmph.biBitCount, 2, 1, file);
    fwrite(&bmph.biCompression, 4, 1, file);
    fwrite(&bmph.biSizeImage, 4, 1, file);
    fwrite(&bmph.biXPelsPerMeter, 4, 1, file);
    fwrite(&bmph.biYPelsPerMeter, 4, 1, file);
    fwrite(&bmph.biClrUsed, 4, 1, file);
    fwrite(&bmph.biClrImportant, 4, 1, file);

    line = (unsigned char *)malloc(bytesPerLine);
    if (line == NULL)
    {
        fprintf(stderr, "Can't allocate memory for BMP file.\n");
        return(0);
    }

    for (i = height - 1; i >= 0; i--)
    {
        for (j = 0; j < width; j++)
        {
            ipos = 3 * (width * i + j);
            line[3*j] = rgb[ipos + 2];
            line[3*j+1] = rgb[ipos + 1];
            line[3*j+2] = rgb[ipos];
        }
        fwrite(line, bytesPerLine, 1, file);
    }

    free(line);
    fclose(file);

    return(1);
}

void mandelbrot(int width, int height, int max_iter)
{
	// Multiply by 3 here, since we need red, green and blue for each pixel
  size_t buffer_size = sizeof(char) * width * height * 3;

	char *image;
  hipMalloc((void **) &image, buffer_size);

  char *host_image = (char *) malloc(buffer_size);

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(width / blockDim.x, height / blockDim.y, 1);
  render<<< gridDim, blockDim, 0 >>>(image, width, height, max_iter);

  hipMemcpy(host_image, image, buffer_size, hipMemcpyDeviceToHost);

  // Now write the file
  write_bmp("output.bmp", width, height, host_image);

  hipFree(image);
  free(host_image);
}
