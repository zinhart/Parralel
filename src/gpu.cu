#include "hip/hip_runtime.h"
#include "concurrent_routines/concurrent_routines.hh"
#include "concurrent_routines/timer.hh"
#include <iostream>
namespace zinhart
{
  // KERNELS
  __global__ void parallel_saxpy_kernel(const float a, float * x, float * y, const std::uint32_t N)
  {
   	std::uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(thread_id > N)
	  return;
	y[thread_id] = a * x[thread_id] + y[thread_id];
  }
  template<class Precision_Type>
  __global__ void naive_matrix_product(const std::uint32_t & N, Precision_Type * A, Precision_Type * B, Precision_Type * C)
  {
  	std::uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(thread_id > N)
	  return;
  }
  
  // GPU WRAPPERS
  HOST int parallel_saxpy_gpu(
  		const float & a, float * x, float * y, const std::uint32_t N)
  {
	hipError_t error_id;
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
  	dim3 block_launch;
	std::int32_t warp_size = properties.warpSize;
	std::int32_t threads_per_block = (N + warp_size -1) / warp_size * warp_size;
	if(threads_per_block > 4 * warp_size)
	  threads_per_block = 4 * warp_size;
	block_launch.x = (N + threads_per_block - 1) / threads_per_block;// number of blocks
	block_launch.y = 1;
	block_launch.z = 1;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	parallel_saxpy_kernel<<<block_launch, threads_per_block >>>(a, x, y, N);
	hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
	error_id = hipGetLastError();
  	if(error_id != hipSuccess)
	{
	  std::cerr<<"saxpy failed to launch with error: "<<hipGetErrorString(error_id)<<"\n";
	  return 1;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"\ngpu time: "<<milliseconds;
	return 0;
  }
}
